
#include <hip/hip_runtime.h>
#define BSIZE 32

__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    int bx = blockIdx.x;    int by = 2*blockIdx.y;
    int tx = threadIdx.x;  int ty = 2*threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*by;
    int bBegin = blockDim.y*bx;
    double Csub0 = 0;
    double Csub1 = 0;
    // Declaration of shared memory buffers 
    __shared__ double As[BSIZE][2*BSIZE];
    __shared__ double Bs[BSIZE][BSIZE];

    // Starting index of A and B for the thread
    int aInd = aBegin + Ni * tx + ty;
    int bInd = bBegin + Nj * threadIdx.y + tx;

    int cBegin = bx*blockDim.x + by*blockDim.y*Nj;
    int cInd = cBegin + ty*Nj + tx;

    int aEnd = Ni*Nk;
    int bEnd = Nj*Nk;
    int cEnd = Ni*Nj;

    for (int kt = 0; kt < Nk; kt += BSIZE) {

        if( aBegin + ty + 1 < Ni && aInd < aEnd ) {
            As[tx][ty] = A[ aInd ];
            As[tx][ty + 1] = A[ aInd + 1 ];
            
        }
        else if( aBegin + ty < Ni && aInd < aEnd ) {
            As[tx][ty] = A[ aInd ];
            As[tx][ty + 1] = 0;
        }
        else {
            As[tx][ty] = 0;
            As[tx][ty + 1] = 0;
        }

        if( bBegin + tx < Nj && bInd < bEnd ) {
            Bs[threadIdx.y][tx] = B[ bInd ];
        }
        else {
            Bs[threadIdx.y][tx] = 0;
        }

        __syncthreads();

        // if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {

        for (int k = kt; k < kt + BSIZE; k++) { 
        
            Csub0 += As[k - kt][ty]*Bs[k - kt][tx];
            Csub1 += As[k - kt][ty + 1] * Bs[k - kt][tx];
        
        }
        // }
        // else if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {

        //     for (int k = kt; k < kt + BSIZE; k++) { 
            
        //         Csub0 += As[k - kt][ty] * Bs[k - kt][tx];
            
        //     }

        // }
        
        __syncthreads();
        aInd += Ni*blockDim.y;
        bInd += Nj*blockDim.y;
    }

    if( bx*blockDim.x + tx < Nj && cInd + Nj < cEnd ) {
        C[ cInd ] = Csub0;
        C[ cInd + Nj ] = Csub1;
    }
    else if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub0;
    }
}