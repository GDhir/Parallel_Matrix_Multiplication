
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int i = 2*blockIdx.y;
        int j = 2*blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i + 1 < Ni ) && ( j + blockDim.x < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
            double sum2 = 0;
            double sum3 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                 sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                 sum1 += A[ k*Ni + i + 1 ]*B[ k*Nj + j ]; 
                 sum2 += A[ k*Ni + i ]*B[ k*Nj + j + blockDim.x ]; 
                 sum3 += A[ k*Ni + i + 1 ]*B[ k*Nj + j + blockDim.x ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ (i + 1) * Nj + j ] = sum1;
            C[ i * Nj + j + blockDim.x ] = sum2;
            C[ (i + 1) * Nj + j + blockDim.x ] = sum3;
        }
        else if( i + 1 < Ni && j < Nj ) {

            double sum0 = 0;
            double sum1 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                 sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                 sum1 += A[ k*Ni + i + 1 ]*B[ k*Nj + j ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ (i + 1) * Nj + j ] = sum1;

        }
        else if( ( i < Ni ) && ( j + blockDim.x < Nj ) ) {

            double sum0 = 0;
            double sum1 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                 sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                 sum1 += A[ k*Ni + i ]*B[ k*Nj + j + blockDim.x ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ i * Nj + j + blockDim.x ] = sum1;

        }
        else if( i < Ni && j < Nj ) {

            double sum = 0;

            for (int k = 0; k < Nk; ++k) {
                sum += A[ k*Ni + i ]*B[ k*Nj + j ]; 
            }
                
            C[ i * Nj + j ] = sum;

        }
    
    }
    
    