
#include <hip/hip_runtime.h>
#define BSIZE 32

__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    int bx = 2*blockIdx.x;    int by = blockIdx.y;
    int tx = threadIdx.x;  int ty = threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*by;
    int bBegin = blockDim.y*bx;
    double Csub0 = 0;
    double Csub1 = 0;
    // Declaration of shared memory buffers 
    __shared__ double As[BSIZE][BSIZE];
    __shared__ double Bs[BSIZE][2*BSIZE];

    // Starting index of A and B for the thread
    int aInd = aBegin + Ni * tx + ty;
    int bInd = bBegin + Nj * ty + tx;

    int cBegin = bx*blockDim.x + by*blockDim.y*Nj;
    int cInd = cBegin + ty*Nj + tx;

    int aEnd = Ni*Nk;
    int bEnd = Nj*Nk;
    int cEnd = Ni*Nj;

    for (int kt = 0; kt < Nk; kt += blockDim.y) {

        if( aBegin + ty < Ni && aInd < aEnd ) {
            As[tx][ty] = A[ aInd ];
            
        }
        else {
            As[tx][ty] = 0;

        }

        if( bBegin + tx + BSIZE < Nj && bInd < bEnd ) {
            Bs[ty][tx] = B[ bInd ];
            Bs[ty][tx + BSIZE] = B[ bInd + BSIZE ];
        }
        else if( bBegin + tx < Nj && bInd < bEnd ) {
            Bs[ty][tx] = B[ bInd ];
            Bs[ty][tx + BSIZE] = 0;
        }
        else {
            Bs[ty][tx] = 0;
            Bs[ty][tx + BSIZE] = 0;
        }

        __syncthreads();

        // if( bx*blockDim.x + tx + BSIZE < Nj && cInd < cEnd ) {

            for (int k = kt; k < kt + BSIZE; k++) { 
            
                Csub0 += As[k - kt][ty] * Bs[k - kt][tx];
                Csub1 += As[k - kt][ty] * Bs[k - kt][tx + BSIZE];
            
            }
        // }
        // else if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {
            
        //     for (int k = kt; k < kt + BSIZE; k++) { 
            
        //         Csub0 += As[k - kt][ty] * Bs[k - kt][tx];
            
        //     }
        // }
        
        __syncthreads();
        aInd += Ni*blockDim.y;
        bInd += Nj*blockDim.y;
    }

    if( bx*blockDim.x + tx + BSIZE < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub0;
        C[ cInd + BSIZE ] = Csub1;
    }
    else if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub0;
    }
}