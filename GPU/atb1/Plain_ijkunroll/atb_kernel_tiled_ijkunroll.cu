
#include <hip/hip_runtime.h>
#define BSIZE 32

__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    int bx = 2*blockIdx.x;    int by = 2*blockIdx.y;
    int tx = threadIdx.x;  int ty = 2*threadIdx.y;
    int tyreal = threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*by;
    int bBegin = blockDim.y*bx;
    double Csub0 = 0;
    double Csub1 = 0;
    double Csub2 = 0;
    double Csub3 = 0;
    // Declaration of shared memory buffers 
    __shared__ double As[BSIZE][2*BSIZE];
    __shared__ double Bs[BSIZE][2*BSIZE];

    // Starting index of A and B for the thread
    int aInd = aBegin + Ni * tx + ty;
    int bInd = bBegin + Nj * tyreal + tx;

    int cBegin = bx*blockDim.x + by*blockDim.y*Nj;
    int cInd = cBegin + ty*Nj + tx;

    int aEnd = Ni*Nk;
    int bEnd = Nj*Nk;
    int cEnd = Ni*Nj;

    int remk;
    int k;

    for (int kt = 0; kt < Nk; kt += BSIZE) {

        if( aBegin + ty + 1 < Ni && aInd < aEnd ) {
            As[tx][ty] = A[ aInd ];
            As[tx][ty + 1] = A[ aInd + 1 ];
            
        }
        else if( aBegin + ty < Ni && aInd < aEnd ) {
            As[tx][ty] = A[ aInd ];
            As[tx][ty + 1] = 0;
        }
        else {
            As[tx][ty] = 0;
            As[tx][ty + 1] = 0;
        }

        if( bBegin + tx + BSIZE < Nj && bInd < bEnd ) {
            Bs[tyreal][tx] = B[ bInd ];
            Bs[tyreal][tx + BSIZE] = B[ bInd + BSIZE ];
        }
        else if( bBegin + tx < Nj && bInd < bEnd ) {
            Bs[tyreal][tx] = B[ bInd ];
            Bs[tyreal][tx + BSIZE] = 0;
        }
        else {
            Bs[tyreal][tx] = 0;
            Bs[tyreal][tx + BSIZE] = 0;
        }

        __syncthreads();

        // if( bx*blockDim.x + tx + BSIZE < Nj && cInd + Nj < cEnd ) {

            remk = BSIZE%2;

            for( k = kt; k < kt + remk; k++ ) {

                Csub0 += As[k - kt][ty] * Bs[k - kt][tx];
                Csub1 += As[k - kt][ty] * Bs[k - kt][tx + BSIZE];
                Csub2 += As[k - kt][ty + 1] * Bs[k - kt][tx];
                Csub3 += As[k - kt][ty + 1] * Bs[k - kt][tx + BSIZE];

            }


            for ( k = kt + remk; k < kt + BSIZE; k += 2) { 
            
                Csub0 += As[k - kt][ty] * Bs[k - kt][tx];
                Csub1 += As[k - kt][ty] * Bs[k - kt][tx + BSIZE];
                Csub2 += As[k - kt][ty + 1] * Bs[k - kt][tx];
                Csub3 += As[k - kt][ty + 1] * Bs[k - kt][tx + BSIZE];

                Csub0 += As[k + 1 - kt][ty] * Bs[k + 1 - kt][tx];
                Csub1 += As[k + 1 - kt][ty] * Bs[k + 1 - kt][tx + BSIZE];
                Csub2 += As[k + 1 - kt][ty + 1] * Bs[k + 1 - kt][tx];
                Csub3 += As[k + 1 - kt][ty + 1] * Bs[k + 1 - kt][tx + BSIZE];
            
            }
        // }
        // else if( bx*blockDim.x + tx + BSIZE < Nj && cInd < cEnd ) {

        //     for (int k = kt; k < kt + BSIZE; k++) { 
            
        //         Csub0 += As[k - kt][ty] * Bs[k - kt][tx];
        //         Csub1 += As[k - kt][ty] * Bs[k - kt][tx + BSIZE];
            
        //     }
        // }
        // else if( bx*blockDim.x + tx < Nj && cInd + Nj < cEnd ) {

        //     for (int k = kt; k < kt + BSIZE; k++) { 
            
        //         Csub0 += As[k - kt][ty] * Bs[k - kt][tx];
        //         Csub2 += As[k - kt][ty + 1] * Bs[k - kt][tx];
            
        //     }

        // }
        // else if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {
            
        //     for (int k = kt; k < kt + BSIZE; k++) { 
            
        //         Csub0 += As[k - kt][ty] * Bs[k - kt][tx];
            
        //     }
        // }
        
        __syncthreads();
        aInd += Ni*blockDim.y;
        bInd += Nj*blockDim.y;
    }

    if( bx*blockDim.x + tx + BSIZE < Nj && cInd + Nj < cEnd ) {
        C[ cInd ] = Csub0;
        C[ cInd + BSIZE ] = Csub1;
        C[ cInd + Nj ] = Csub2;
        C[ cInd + BSIZE + Nj ] = Csub3;
    }
    else if( bx*blockDim.x + tx + BSIZE < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub0;
        C[ cInd + BSIZE ] = Csub1;
    }
    else if( bx*blockDim.x + tx < Nj && cInd + Nj < cEnd ) {
        C[ cInd ] = Csub0;
        C[ cInd + Nj ] = Csub2;
    }
    else if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub0;
    }
}