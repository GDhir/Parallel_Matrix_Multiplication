
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int i = 2*( blockIdx.y*blockDim.y+threadIdx.y );
        int j = 2*blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i + 1 < Ni ) && ( j + blockDim.x < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
            double sum2 = 0;
            double sum3 = 0;

            int remk = Nk%2;

            for( int k = 0; k < remk; k++ ) {
                sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                sum1 += A[ k*Ni + i + 1 ]*B[ k*Nj + j ]; 
                sum2 += A[ k*Ni + i ]*B[ k*Nj + j + blockDim.x ]; 
                sum3 += A[ k*Ni + i + 1 ]*B[ k*Nj + j + blockDim.x ]; 
            }
    
            for (int k = remk; k < Nk; k += 2) {
                sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                sum1 += A[ k*Ni + i + 1 ]*B[ k*Nj + j ]; 
                sum2 += A[ k*Ni + i ]*B[ k*Nj + j + blockDim.x ]; 
                sum3 += A[ k*Ni + i + 1 ]*B[ k*Nj + j + blockDim.x ]; 

                sum0 += A[ ( k + 1 )*Ni + i ]*B[ (k + 1)*Nj + j ]; 
                sum1 += A[ ( k + 1 )*Ni + i + 1 ]*B[ (k + 1)*Nj + j ]; 
                sum2 += A[ ( k + 1 )*Ni + i ]*B[ (k + 1)*Nj + j + blockDim.x ]; 
                sum3 += A[ ( k + 1 )*Ni + i + 1 ]*B[ (k + 1)*Nj + j + blockDim.x ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ (i + 1) * Nj + j ] = sum1;
            C[ i * Nj + j + blockDim.x ] = sum2;
            C[ (i + 1) * Nj + j + blockDim.x ] = sum3;
        }
        else if( ( i + 1 < Ni ) && ( j < Nj ) ) {

            double sum0 = 0;
            double sum1 = 0;

            int remk = Nk%2;

            for( int k = 0; k < remk; k++ ) {
                sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                sum1 += A[ k*Ni + i + 1 ]*B[ k*Nj + j ]; 
            }
    
            for (int k = remk; k < Nk; k += 2) {
                sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                sum1 += A[ k*Ni + i + 1 ]*B[ k*Nj + j ]; 

                sum0 += A[ ( k + 1 )*Ni + i ]*B[ (k + 1)*Nj + j ]; 
                sum1 += A[ ( k + 1 )*Ni + i + 1 ]*B[ (k + 1)*Nj + j ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ (i + 1) * Nj + j ] = sum1;
        }
        else if( ( i < Ni ) && ( j + blockDim.x < Nj ) ) {

            double sum0 = 0;
            double sum1 = 0;
            
            int remk = Nk%2;

            for( int k = 0; k < remk; k++ ) {
                sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                sum1 += A[ k*Ni + i ]*B[ k*Nj + j + blockDim.x ]; 
            }
    
            for (int k = remk; k < Nk; k += 2) {
                sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                sum1 += A[ k*Ni + i ]*B[ k*Nj + j + blockDim.x ]; 

                sum0 += A[ ( k + 1 )*Ni + i ]*B[ (k + 1)*Nj + j ]; 
                sum1 += A[ ( k + 1 )*Ni + i ]*B[ (k + 1)*Nj + j + blockDim.x ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ i * Nj + j + blockDim.x ] = sum1;

        }
        else if( ( i < Ni ) && ( j < Nj ) ) {

            double sum0 = 0;
            
            int remk = Nk%2;

            for( int k = 0; k < remk; k++ ) {
                sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
            }
    
            for (int k = remk; k < Nk; k += 2) {
                sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                sum0 += A[ ( k + 1 )*Ni + i ]*B[ (k + 1)*Nj + j ]; 
            }
                 
            C[ i * Nj + j ] = sum0;

        }
    
    }
    
    