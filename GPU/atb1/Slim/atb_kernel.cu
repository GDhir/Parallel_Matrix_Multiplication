
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
// Initially empty; will clearly not pass correctness

    int i = blockIdx.y;
    int j = blockIdx.x*blockDim.x+threadIdx.x;
    
    if ( ( i < Ni ) && ( j < Nj ) ) { 
        double sum = 0;

        for (int k = 0; k < Nk; ++k) {
             sum += A[ k*Ni + i ]*B[ k*Nj + j ]; 
        }
             
        C[ i * Nj + j ] = sum;
    }

}

