
#include <hip/hip_runtime.h>
#define BSIZE 32

// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness

        int tx = threadIdx.x;
        int i = blockIdx.y;
        int j = blockIdx.x*blockDim.x+tx;
        int idx = tx*Ni + i;

        int aEnd = Nk*Ni;
        
        __shared__ double As[BSIZE];

        // if ( ( i < Ni ) && ( j < Nj ) ) { 
            double sum = 0;
    
            for( int ks = 0; ks < Nk; ks += BSIZE ) {

                if( i < Ni && idx + ks*Ni < aEnd )
                    As[tx] = A[ ks*Ni + idx ];
                else
                    As[tx] = 0;

                __syncthreads();

                if( j < Nj ) {
                    for (int k = ks; k < min(Nk, ks + BSIZE); ++k) {
                        sum += As[k - ks]*B[ k*Nj + j ]; 
                    }
                }

                __syncthreads();

            }

            if( i < Ni && j < Nj )
                C[ i * Nj + j ] = sum;
        // }
    
    }
    
    