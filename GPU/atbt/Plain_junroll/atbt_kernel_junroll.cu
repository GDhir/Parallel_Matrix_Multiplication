
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int j = 2*( blockIdx.y*blockDim.y+threadIdx.y );
        int i = blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i < Ni ) && ( j < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
    
            if( j + 1 < Nj ) {
                for (int k = 0; k < Nk; ++k) {
                    sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                    sum1 += A[ k*Ni + i ]*B[ (j + 1)*Nk + k ]; 
                }
            }
            else {
                for (int k = 0; k < Nk; ++k) {
                    sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                }
            }
                 
            C[ i * Nj + j ] = sum0;

            if( j + 1 < Nj )
                C[ i * Nj + j + 1 ] = sum1;
        }
    
    }
    
    