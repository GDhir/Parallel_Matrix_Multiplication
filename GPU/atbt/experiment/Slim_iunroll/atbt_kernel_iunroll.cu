
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int j = blockIdx.y;
        int i = 2*blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i + blockDim.x < Ni ) && ( j < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                sum1 += A[ k*Ni + i + blockDim.x ]*B[ j*Nk + k ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ (i + blockDim.x) * Nj + j ] = sum1;
        }
        else if( ( i < Ni ) && ( j < Nj ) ) {
            double sum0 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
        }
    
    }
    
    