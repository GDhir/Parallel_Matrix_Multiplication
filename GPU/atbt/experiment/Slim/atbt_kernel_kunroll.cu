
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int j = blockIdx.y;
        int i = blockIdx.x*blockDim.x+threadIdx.x;
        
        int remk = Nk%4;

        if ( ( i < Ni ) && ( j < Nj ) ) { 
            double sum = 0;
    
            for( int k = 0; k < remk; k++ ) {
                sum += A[ k*Ni + i ]*B[ j*Nk + k ]; 
            }

            for (int k = remk; k < Nk; k += 4) {
                sum += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                sum += A[ (k + 1)*Ni + i ]*B[ j*Nk + k + 1 ]; 
                sum += A[ (k + 2)*Ni + i ]*B[ j*Nk + k + 2 ]; 
                sum += A[ (k + 3)*Ni + i ]*B[ j*Nk + k + 3 ]; 

            }
                 
            C[ i * Nj + j ] = sum;
        }
    
    }
    
    