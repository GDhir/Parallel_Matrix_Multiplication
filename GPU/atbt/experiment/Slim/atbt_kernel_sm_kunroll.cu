
#include <hip/hip_runtime.h>
#define BSIZE 32

// matrix multiply kernel: C = A^T * B^T
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness

        int tx = threadIdx.x;
        int j = blockIdx.y;
        int i = blockIdx.x*blockDim.x+tx;
        int idx = j*Nk + tx;
        
        __shared__ double Bs[BSIZE];

        int kend = Nk*(j + 1);

        double sum = 0;
        int remk = BSIZE%2;

        for( int ks = 0; ks < Nk; ks += BSIZE ) {

            if( j < Nj && idx + ks < kend )
                Bs[tx] = B[ idx + ks ];
            else
                Bs[tx] = 0;

            __syncthreads();

            if( i < Ni ) {

                for( int k = ks; k < ks + remk; k++ ) {
                    sum += A[k*Ni + i]*Bs[ k - ks ]; 
                }

                for (int k = ks + remk; k < ks + BSIZE; k += 2) {
                    sum += A[k*Ni + i]*Bs[ k - ks ]; 
                    sum += A[(k + 1)*Ni + i]*Bs[ k + 1 - ks ]; 
                }
            }

            __syncthreads();

        }

        if( i < Ni && j < Nj )
            C[ i * Nj + j ] = sum;
    
    }
    
    