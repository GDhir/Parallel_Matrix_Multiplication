
#include <hip/hip_runtime.h>
#define BSIZE 32

__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    int bx = 2*blockIdx.x;    int by = blockIdx.y;
    int tx = threadIdx.x;  int ty = threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*bx;
    int bBegin = blockDim.y*by*Nk;
    double Csub0 = 0;
    double Csub1 = 0;
    // Declaration of shared memory buffers 
    __shared__ double As[BSIZE][2*BSIZE];
    __shared__ double Bs[BSIZE][BSIZE];

    // Starting index of A and B for the thread
    int aInd = aBegin + Ni * ty + tx;
    int bInd = bBegin + Nk * ty + tx;

    int cBegin = bx*blockDim.x*Nj + by*blockDim.y;
    int cInd = cBegin + tx*Nj + ty;

    int aEnd = Ni*Nk;
    int bEnd = Nj*Nk;
    int cEnd = Ni*Nj;

    int rowend = ( blockDim.y*by + ty + 1 )*Nk;

    for (int kt = 0; kt < Nk; kt += BSIZE) {

        if( aBegin + tx + BSIZE < Ni && aInd < aEnd ) {
            As[ty][tx + BSIZE] = A[ aInd + BSIZE ];
            As[ty][tx] = A[ aInd ];
            
        }
        else if( aBegin + tx < Ni && aInd < aEnd ) {
            As[ty][tx] = A[ aInd ];
            As[ty][tx + BSIZE] = 0;
        }
        else {
            As[ty][tx] = 0;
            As[ty][tx + BSIZE] = 0;
        }

        if( bInd < rowend && bInd < bEnd ) {
            Bs[ty][tx] = B[ bInd ];
        }
        else {
            Bs[ty][tx] = 0;
        }

        __syncthreads();

        for (int k = kt; k < kt + blockDim.y; ++k) { 
        
            Csub0 += As[k - kt][tx] * Bs[ty][k - kt];
            Csub1 += As[k - kt][tx + BSIZE] * Bs[ty][k - kt];
        
        }
        
        __syncthreads();
        aInd += Ni*blockDim.y;
        bInd += BSIZE;
    }

    if( by*blockDim.y + ty < Nj && cInd + BSIZE*Nj < cEnd ) {
        C[ cInd ] = Csub0;
        C[ cInd + BSIZE*Nj ] = Csub1;
    }
    else if( by*blockDim.y + ty < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub0;
    }
}