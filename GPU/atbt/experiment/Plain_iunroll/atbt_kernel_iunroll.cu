
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int j = blockIdx.y*blockDim.y+threadIdx.y;
        int i = 2*blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i < Ni ) && ( j < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
    
            if( i + blockDim.x < Ni ) {
                for (int k = 0; k < Nk; ++k) {
                    sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                    sum1 += A[ k*Ni + i + blockDim.x ]*B[ j*Nk + k ]; 
                }
            }
            else {

                for (int k = 0; k < Nk; ++k) {
                    sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                }

            }

            C[ i * Nj + j ] = sum0;

            if( i + blockDim.x < Ni ) {
                C[ (i + blockDim.x) * Nj + j ] = sum1;
            }
        }
    
    }
    
    