
#include <hip/hip_runtime.h>
#define BSIZE 32

__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    int bx = 2*blockIdx.x;    int by = 2*blockIdx.y;
    int tx = threadIdx.x;  int ty = 2*threadIdx.y;
    int tyreal = threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*bx;
    int bBegin = blockDim.y*by*Nk;
    double Csub0 = 0;
    double Csub1 = 0;
    double Csub2 = 0;
    double Csub3 = 0;
    // Declaration of shared memory buffers 
    __shared__ double As[BSIZE][2*BSIZE];
    __shared__ double Bs[2*BSIZE][BSIZE];

    // Starting index of A and B for the thread
    int aInd = aBegin + Ni * tyreal + tx;
    int bInd = bBegin + Nk * ty + tx;

    int cBegin = bx*blockDim.x*Nj + by*blockDim.y;
    int cInd = cBegin + tx*Nj + ty;

    int aEnd = Ni*Nk;
    int bEnd = Nj*Nk;
    int cEnd = Ni*Nj;

    int remk = BSIZE%2;

    int rowend = ( blockDim.y*by + ty + 1 )*Nk;

    for (int kt = 0; kt < Nk; kt += BSIZE) {

        if( aBegin + tx + BSIZE < Ni && aInd < aEnd ) {
            As[tyreal][tx + BSIZE] = A[ aInd + BSIZE ];
            As[tyreal][tx] = A[ aInd ];
            
        }
        else if( aBegin + tx < Ni && aInd < aEnd ) {
            As[tyreal][tx] = A[ aInd ];
            As[tyreal][tx + BSIZE] = 0;
        }
        else {
            As[tyreal][tx] = 0;
            As[tyreal][tx + BSIZE] = 0;
        }


        if( bInd < rowend && bInd + Nk < bEnd ) {
            Bs[ty + 1][tx] = B[ bInd + Nk ];
            Bs[ty][tx] = B[ bInd ];
        }
        else if( bInd < rowend && bInd < bEnd ) {
            Bs[ty + 1][tx] = 0;
            Bs[ty][tx] = B[bInd];
        }
        else {
            Bs[ty + 1][tx] = 0;
            Bs[ty][tx] = 0;
        }

        __syncthreads();

        for( int k = kt; k < kt + remk; k++ ) {
            Csub0 += As[k - kt][tx] * Bs[ty][k - kt];
            Csub1 += As[k - kt][tx + BSIZE] * Bs[ty][k - kt];
            Csub2 += As[k - kt][tx] * Bs[ty + 1][k - kt];
            Csub3 += As[k - kt][tx + BSIZE] * Bs[ty + 1][k - kt];
        }

        for (int k = kt + remk; k < kt + BSIZE; k += 2) { 
        
            Csub0 += As[k - kt][tx] * Bs[ty][k - kt];
            Csub1 += As[k - kt][tx + BSIZE] * Bs[ty][k - kt];
            Csub2 += As[k - kt][tx] * Bs[ty + 1][k - kt];
            Csub3 += As[k - kt][tx + BSIZE] * Bs[ty + 1][k - kt];

            Csub0 += As[k + 1 - kt][tx] * Bs[ty][k + 1 - kt];
            Csub1 += As[k + 1 - kt][tx + BSIZE] * Bs[ty][k + 1 - kt];
            Csub2 += As[k + 1 - kt][tx] * Bs[ty + 1][k + 1 - kt];
            Csub3 += As[k + 1 - kt][tx + BSIZE] * Bs[ty + 1][k + 1 - kt];
        
        }
        
        __syncthreads();
        aInd += Ni*BSIZE;
        bInd += BSIZE;
    }

    if( by*blockDim.y + ty + 1 < Nj && cInd + Nj < cEnd ) {
        C[ cInd ] = Csub0;
        C[ cInd + Nj*BSIZE ] = Csub1;
        C[ cInd + 1 ] = Csub2;
        C[ cInd + Nj*BSIZE + 1 ] = Csub3;
    }
    else if( by*blockDim.y + ty + 1 < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub0;
        C[ cInd + 1 ] = Csub2;
    }
    else if( by*blockDim.y + ty < Nj && cInd + Nj < cEnd ) {
        C[ cInd ] = Csub0;
        C[ cInd + Nj*BSIZE ] = Csub1;
    }
    else if( by*blockDim.y + ty < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub0;
    }
}