
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int j = blockIdx.y*blockDim.y+threadIdx.y;
        int i = blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i < Ni ) && ( j < Nj ) ) { 
            double sum = 0;
    
            int remk = Nk%2;
            
            for( int k = 0; k < remk; k++ ) {
                sum += A[ k*Ni + i ]*B[ j*Nk + k ]; 
            }

            for (int k = remk; k < Nk; k += 2) {
                 sum += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                 sum += A[ (k + 1)*Ni + i ]*B[ j*Nk + k + 1 ]; 
            }
                 
            C[ i * Nj + j ] = sum;
        }
    
    }
    
    