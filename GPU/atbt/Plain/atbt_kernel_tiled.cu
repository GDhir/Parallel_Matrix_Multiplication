
#include <hip/hip_runtime.h>
#define BSIZE 32

__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    int bx = blockIdx.x;    int by = blockIdx.y;
    int tx = threadIdx.x;  int ty = threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*bx;
    int bBegin = blockDim.y*by*Nk;
    double Csub = 0;
    // Declaration of shared memory buffers 
    __shared__ double As[BSIZE][BSIZE];
    __shared__ double Bs[BSIZE][BSIZE];

    // Starting index of A and B for the thread
    int aInd = aBegin + Ni * ty + tx;
    int bInd = bBegin + Nk * ty + tx;

    int cBegin = bx*blockDim.x*Nj + by*blockDim.y;
    int cInd = cBegin + tx*Nj + ty;

    int aEnd = Ni*Nk;
    int bEnd = Nj*Nk;
    int cEnd = Ni*Nj;

    int rowend = ( blockDim.y*by + ty + 1 )*Nk;

    for (int kt = 0; kt < Nk; kt += BSIZE) {

        if( aBegin + tx < Ni && aInd < aEnd ) {
            As[ty][tx] = A[ aInd ];
            
        }
        else {
            As[ty][tx] = 0;

        }

        if( bInd < rowend && bInd < bEnd ) {
            Bs[ty][tx] = B[ bInd ];
        }
        else {
            Bs[ty][tx] = 0;
        }

        __syncthreads();

        for (int k = kt; k < kt + blockDim.y; ++k) { 
        
            Csub += As[k - kt][tx] * Bs[ty][k - kt];
        
        }
        
        __syncthreads();
        aInd += Ni*blockDim.y;
        bInd += BSIZE;
    }

    if( by*blockDim.y + ty < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub;
    }
}