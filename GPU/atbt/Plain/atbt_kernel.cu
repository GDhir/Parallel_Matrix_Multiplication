
#include <hip/hip_runtime.h>
 // matrix multiply kernel: C = A^T * B^T
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; clearly Will not pass correctness test
    
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if( i < Ni && j < Nj ) {

        double csub = 0;
    
        for( int k = 0; k < Nk; k++ ) {
            csub += A[ k*Ni + i ]*B[ j*Nk + k ];
        }

        C[ i*Nj + j ] = csub;
    }
        
    
}
    