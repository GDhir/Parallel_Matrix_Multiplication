
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int j = 2*blockIdx.y;
        int i = blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i < Ni ) && ( j + 1 < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                sum1 += A[ k*Ni + i ]*B[ (j + 1)*Nk + k ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ i * Nj + j + 1 ] = sum1;
        }
        else if( i < Ni && j < Nj ) {

            double sum0 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
            }
                    
            C[ i * Nj + j ] = sum0;

        }
    
    }
    
    