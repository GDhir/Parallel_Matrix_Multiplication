
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int j = 2*blockIdx.y;
        int i = 2*blockIdx.x*blockDim.x+threadIdx.x;
        int remk = Nk%2;
        if ( ( i + blockDim.x < Ni ) && ( j + 1 < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
            double sum2 = 0;
            double sum3 = 0;

            for( int k = 0; k < remk; k++ ) {
                sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                sum1 += A[ k*Ni + i + blockDim.x ]*B[ j*Nk + k ]; 
                sum2 += A[ k*Ni + i ]*B[ (j + 1)*Nk + k ]; 
                sum3 += A[ k*Ni + i + blockDim.x ]*B[ (j + 1)*Nk + k ];
            }
    
            for (int k = remk; k < Nk; k += 2) {
                sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                sum1 += A[ k*Ni + i + blockDim.x ]*B[ j*Nk + k ]; 
                sum2 += A[ k*Ni + i ]*B[ (j + 1)*Nk + k ]; 
                sum3 += A[ k*Ni + i + blockDim.x ]*B[ (j + 1)*Nk + k ]; 

                sum0 += A[ (k + 1)*Ni + i ]*B[ j*Nk + k + 1 ]; 
                sum1 += A[ (k + 1)*Ni + i + blockDim.x ]*B[ j*Nk + k + 1 ]; 
                sum2 += A[ (k + 1)*Ni + i ]*B[ (j + 1)*Nk + k + 1 ]; 
                sum3 += A[ (k + 1)*Ni + i + blockDim.x ]*B[ (j + 1)*Nk + k + 1 ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ (i + blockDim.x) * Nj + j ] = sum1;
            C[ i * Nj + j + 1 ] = sum2;
            C[ (i + blockDim.x) * Nj + j + 1 ] = sum3;
        }
        else if( i + blockDim.x < Ni && j < Nj ) {

            double sum0 = 0;
            double sum1 = 0;

            for( int k = 0; k < remk; k++ ) {
                sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                sum1 += A[ k*Ni + i + blockDim.x ]*B[ j*Nk + k ]; 
            }
    
            for (int k = remk; k < Nk; k += 2) {
                 sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                 sum1 += A[ k*Ni + i + blockDim.x ]*B[ j*Nk + k ]; 

                 sum0 += A[ (k + 1)*Ni + i ]*B[ j*Nk + (k + 1) ]; 
                 sum1 += A[ (k + 1)*Ni + i + blockDim.x ]*B[ j*Nk + (k + 1) ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ (i + blockDim.x) * Nj + j ] = sum1;

        }
        else if( ( i < Ni ) && ( j + 1 < Nj ) ) {

            double sum0 = 0;
            double sum1 = 0;
    
            for( int k = 0; k < remk; k++ ) {
                sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                sum1 += A[ k*Ni + i ]*B[ (j + 1)*Nk + k ]; 
            }

            for (int k = remk; k < Nk; k += 2) {
                 sum0 += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                 sum1 += A[ k*Ni + i ]*B[ (j + 1)*Nk + k ]; 

                 sum0 += A[ (k + 1)*Ni + i ]*B[ j*Nk + k + 1 ]; 
                 sum1 += A[ (k + 1)*Ni + i ]*B[ (j + 1)*Nk + k + 1 ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ i * Nj + j + 1 ] = sum1;

        }
        else if( i < Ni && j < Nj ) {

            double sum = 0;

            for( int k = 0; k < remk; k++ ) {
                sum += A[ k*Ni + i ]*B[ j*Nk + k ]; 
            }

            for (int k = remk; k < Nk; k += 2) {
                sum += A[ k*Ni + i ]*B[ j*Nk + k ]; 
                sum += A[ (k + 1)*Ni + i ]*B[ j*Nk + k + 1 ]; 
            }
                
            C[ i * Nj + j ] = sum;

        }
    
    }
    
    