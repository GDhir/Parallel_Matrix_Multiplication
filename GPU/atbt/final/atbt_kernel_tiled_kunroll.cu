
#include <hip/hip_runtime.h>
#define BSIZE 32

__global__ void atbt_kernel_tiled_kunroll(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    int bx = blockIdx.x;    int by = blockIdx.y;
    int tx = threadIdx.x;  int ty = threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*bx;
    int bBegin = blockDim.y*by*Nk;
    double Csub = 0;
    // Declaration of shared memory buffers 
    __shared__ double As[BSIZE][BSIZE];
    __shared__ double Bs[BSIZE][BSIZE];

    // Starting index of A and B for the thread
    int aInd = aBegin + Ni * ty + tx;
    int bInd = bBegin + Nk * ty + tx;

    int cBegin = bx*blockDim.x*Nj + by*blockDim.y;
    int cInd = cBegin + tx*Nj + ty;

    int aEnd = Ni*Nk;
    int bEnd = Nj*Nk;
    int cEnd = Ni*Nj;

    int remk = BSIZE%2;

    int rowend = ( blockDim.y*by + ty + 1 )*Nk;

    for (int kt = 0; kt < Nk; kt += BSIZE) {

        if( aBegin + tx < Ni && aInd < aEnd ) {
            As[ty][tx] = A[ aInd ];
            
        }
        else {
            As[ty][tx] = 0;

        }

        if( bInd < rowend && bInd < bEnd ) {
            Bs[ty][tx] = B[ bInd ];
        }
        else {
            Bs[ty][tx] = 0;
        }

        __syncthreads();

        for( int k = kt; k < kt + remk; k++ ) {
            Csub += As[k - kt][tx] * Bs[ty][k - kt];
        }

        for (int k = kt + remk; k < kt + blockDim.y; k += 2) { 
        
            Csub += As[k - kt][tx] * Bs[ty][k - kt];
            Csub += As[k + 1 - kt][tx] * Bs[ty][k + 1 - kt];
        
        }
        
        __syncthreads();
        aInd += Ni*blockDim.y;
        bInd += BSIZE;
    }

    if( by*blockDim.y + ty < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub;
    }
}