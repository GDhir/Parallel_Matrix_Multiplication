
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atbt_kernel_slim(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
// Initially empty; will clearly not pass correctness

    int j = blockIdx.y;
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    
    if ( ( i < Ni ) && ( j < Nj ) ) { 
        double sum = 0;

        for (int k = 0; k < Nk; ++k) {
             sum += A[ k*Ni + i ]*B[ j*Nk + k ]; 
        }
             
        C[ i * Nj + j ] = sum;
    }

}

