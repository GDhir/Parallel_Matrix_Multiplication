
#include <hip/hip_runtime.h>
#define BSIZE 16

// matrix multiply kernel: C = A^T * B^T
__global__ void atbt_kernel_slimsm16(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness

        int tx = threadIdx.x;
        int j = blockIdx.y;
        int i = blockIdx.x*blockDim.x+tx;
        int idx = j*Nk + tx;
        
        __shared__ double Bs[BSIZE];

        int kend = Nk*(j + 1);

        double sum = 0;

        for( int ks = 0; ks < Nk; ks += BSIZE ) {

            if( j < Nj && idx + ks < kend )
                Bs[tx] = B[ idx + ks ];
            else
                Bs[tx] = 0;

            __syncthreads();

            if( i < Ni ) {
                for (int k = ks; k < ks + BSIZE; ++k) {
                    sum += A[k*Ni + i]*Bs[ k - ks ]; 
                }
            }

            __syncthreads();

        }

        if( i < Ni && j < Nj )
            C[ i * Nj + j ] = sum;
    
    }
    
    