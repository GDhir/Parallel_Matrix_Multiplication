
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define threshold 0.0000001

#define FIXME1 1
#define FIXME2 2
#define FIXME3 3
#define FIXME4 4

void checkCUDAError(const char *msg);

hipEvent_t start, stop;
float tstart, elapsedTime;

// __global__ void atbt(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atbt_kernel_slim(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atbt_kernel_slimsm32(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atbt_kernel_tiled_ijkunroll(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atbt_kernel_tiled_junroll(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atbt_kernel_tiled_kunroll(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);

void launcher(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

  if( Ni >= 512 && Nj >= 512 && Nk >= 512 ) {

    int BSIZE = 32;
    int bsx = BSIZE;
    int bsy = BSIZE;

    dim3 block(bsx, bsy);
    dim3 grid( ceil( Ni/( (double)2*bsx ) ), ceil( Nj/( (double)2*bsy ) ) );

    atbt_kernel_tiled_ijkunroll<<<grid, block>>>(A, B, C, Ni,Nj,Nk);

  }
  else {

    int BSIZE = 32;
    int bsx = BSIZE;
    int bsy = 1;

    dim3 block(bsx, bsy);
    dim3 grid( ceil( Ni/( (double)bsx ) ), ceil( Nj/( (double)bsy ) ) );

    atbt_kernel_slimsm32<<<grid, block>>>(A, B, C, Ni,Nj,Nk);

  }
  // else {

  //   int BSIZE = 32;
  //   int bsx = BSIZE;
  //   int bsy = 1;

  //   dim3 block(bsx, bsy);
  //   dim3 grid( ceil( Ni/( (double)bsx ) ), ceil( Nj/( (double)bsy ) ) );

  //   atbt_kernel_slim<<<grid, block>>>(A, B, C, Ni,Nj,Nk);

  // }

}

int main(){

  double *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k,Ni,Nj,Nk;

  printf("Specify Matrix dimension Ni, Nj, Nk: ");
  scanf("%d %d %d", &Ni,&Nj,&Nk);
  h_A = (double *) malloc(sizeof(double)*Ni*Nk);
  h_B = (double *) malloc(sizeof(double)*Nk*Nj);
  h_C = (double *) malloc(sizeof(double)*Ni*Nj);
  h_Cref = (double *) malloc(sizeof(double)*Ni*Nj);
  for (i=0; i<Ni; i++)
   for (k=0; k<Nk; k++)
    h_A[k*Ni+i] = k*Ni+i-1;
  for (k=0; k<Nk; k++)
   for (j=0; j<Nj; j++)
    h_B[j*Nk+k] = j*Nk+k+1;
  for (i=0; i<Ni; i++)
   for (j=0; j<Nj; j++) {
    h_C[i*Nj+j] = 0;
    h_Cref[i*Nj+j] = 0;}

  for (i=0;i<Ni;i++)
   for (k=0;k<Nk;k++)
    for (j=0;j<Nj;j++)
  // h_Cref[i][j] += h_A[k][i]*h_B[j][k];
     h_Cref[i*Nj+j] += h_A[i+Ni*k]*h_B[k+Nk*j];
  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, Nk*Ni*sizeof(double));
  hipMalloc(&d_B, Nj*Nk*sizeof(double));
  hipMalloc(&d_C, Ni*Nj*sizeof(double));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, Nk*Ni*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Nj*Nk*sizeof(double), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D failure");

  dim3 block(FIXME1,FIXME2);  
  dim3 grid(FIXME3,FIXME4);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  for(int trial=0;trial<5;trial++)
  {
   hipEventRecord(start);
   // Launch kernel
   launcher( d_A, d_B, d_C, Ni, Nj, Nk );
  //  atbt<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);
   hipEventRecord(stop);
   checkCUDAError("kernel launch");
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start,stop);
//   cudaDeviceSynchronize();
   // Copy results back to host
   hipMemcpy(h_C, d_C, Ni*Nj*sizeof(double), hipMemcpyDeviceToHost);
   checkCUDAError("hipMemcpy D2H");
   for (int l = 0; l < Ni*Nj; l++) if (fabs((h_C[l] - h_Cref[l])/h_Cref[l])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", l, h_C[l], h_Cref[l]); return -1;}
   printf("<Ni=%d,Nj=%d,Nk=%d>: Trial %d: GFLOPS: %.2f\n",Ni,Nj,Nk,trial,2.0e-6*Ni*Nj*Nk/elapsedTime);
  }
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

