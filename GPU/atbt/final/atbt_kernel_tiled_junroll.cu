
#include <hip/hip_runtime.h>
#define BSIZE 32

__global__ void atbt_kernel_tiled_junroll(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    int bx = blockIdx.x;    int by = 2*blockIdx.y;
    int tx = threadIdx.x;  int ty = 2*threadIdx.y;
    int tyreal = threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*bx;
    int bBegin = blockDim.y*by*Nk;
    double Csub0 = 0;
    double Csub1 = 0;
    // Declaration of shared memory buffers 
    __shared__ double As[BSIZE][BSIZE];
    __shared__ double Bs[2*BSIZE][BSIZE];

    // Starting index of A and B for the thread
    int aInd = aBegin + Ni * tyreal + tx;
    int bInd = bBegin + Nk * ty + tx;

    int cBegin = bx*blockDim.x*Nj + by*blockDim.y;
    int cInd = cBegin + tx*Nj + ty;

    int aEnd = Ni*Nk;
    int bEnd = Nj*Nk;
    int cEnd = Ni*Nj;

    int rowend = ( blockDim.y*by + ty + 1 )*Nk;

    for (int kt = 0; kt < Nk; kt += BSIZE) {

        if( aBegin + tx < Ni && aInd < aEnd ) {
            As[tyreal][tx] = A[ aInd ];
            
        }
        else {
            As[tyreal][tx] = 0;

        }

        if( bInd < rowend && bInd + Nk < bEnd ) {
            Bs[ty + 1][tx] = B[ bInd + Nk ];
            Bs[ty][tx] = B[ bInd ];
        }
        else if( bInd < rowend && bInd < bEnd ) {
            Bs[ty + 1][tx] = 0;
            Bs[ty][tx] = B[bInd];
        }
        else {
            Bs[ty + 1][tx] = 0;
            Bs[ty][tx] = 0;
        }

        __syncthreads();

        for (int k = kt; k < kt + blockDim.y; ++k) { 
        
            Csub0 += As[k - kt][tx] * Bs[ty][k - kt];
            Csub1 += As[k - kt][tx] * Bs[ty + 1][k - kt];
        
        }
        
        __syncthreads();
        aInd += Ni*blockDim.y;
        bInd += BSIZE;
    }

    if( by*blockDim.y + ty + 1 < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub0;
        C[ cInd + 1 ] = Csub1;
    }
    else if( by*blockDim.y + ty < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub0;
    }
}