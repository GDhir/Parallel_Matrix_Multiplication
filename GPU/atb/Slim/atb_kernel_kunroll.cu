
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int i = blockIdx.y;
        int j = blockIdx.x*blockDim.x+threadIdx.x;
        
        int remk = Nk%4;

        if ( ( i < Ni ) && ( j < Nj ) ) { 
            double sum = 0;
    
            for( int k = 0; k < remk; k++ ) {
                sum += A[ k*Ni + i ]*B[ k*Nj + j ]; 
            }

            for (int k = remk; k < Nk; k += 4) {
                 sum += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                 sum += A[ (k + 1)*Ni + i ]*B[ (k + 1)*Nj + j ]; 
                 sum += A[ (k + 2)*Ni + i ]*B[ (k + 2)*Nj + j ]; 
                 sum += A[ (k + 3)*Ni + i ]*B[ (k + 3)*Nj + j ]; 


            }
                 
            C[ i * Nj + j ] = sum;
        }
    
    }
    
    