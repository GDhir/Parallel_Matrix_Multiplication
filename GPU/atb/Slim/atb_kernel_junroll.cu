
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int i = blockIdx.y;
        int j = 2*blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i < Ni ) && ( j + blockDim.x < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                 sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                 sum1 += A[ k*Ni + i ]*B[ k*Nj + j + blockDim.x ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ i * Nj + j + blockDim.x ] = sum1;
        }
        else if( ( i < Ni ) && ( j < Nj ) ) {
            double sum0 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                 sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
        }
    
    }
    
    