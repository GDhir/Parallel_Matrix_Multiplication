
#include <hip/hip_runtime.h>
#define BSIZE 32

// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness

        int tx = threadIdx.x;
        int i = blockIdx.y;
        int j = blockIdx.x*blockDim.x+tx;
        int idx = tx*Ni + i;
        int remk;

        int aEnd = Nk*Ni;
        
        __shared__ double As[BSIZE];

        if ( ( i < Ni ) && ( j < Nj ) ) { 
            double sum = 0;
    
            for( int ks = 0; ks < Nk; ks += BSIZE ) {

                if( idx + ks*Ni < aEnd )
                    As[tx] = A[ ks*Ni + idx ];
                else
                    As[tx] = 0;

                __syncthreads();

                remk = BSIZE%2;

                for(int k = ks; k < ks + remk; k++ ) {
                    sum += As[k - ks]*B[ k*Nj + j ];
                }

                for (int k = ks + remk; k < ks + remk + BSIZE; k += 2) {
                    sum += As[k - ks]*B[ k*Nj + j ];
                    sum += As[k + 1 - ks]*B[ (k + 1)*Nj + j ]; 
                }

                __syncthreads();

            }
                 
            C[ i * Nj + j ] = sum;
        }
    
    }
    
    