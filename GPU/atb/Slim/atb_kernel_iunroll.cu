
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int i = 2*blockIdx.y;
        int j = blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i + 1 < Ni ) && ( j < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                 sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                 sum1 += A[ k*Ni + i + 1 ]*B[ k*Nj + j ]; 
            }
                 
            C[ i * Nj + j ] = sum0;
            C[ (i + 1) * Nj + j ] = sum1;
        }
        else if( i < Ni && j < Nj ) {

            double sum0 = 0;
    
            for (int k = 0; k < Nk; ++k) {
                sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
            }
                    
            C[ i * Nj + j ] = sum0;

        }
    
    }
    
    