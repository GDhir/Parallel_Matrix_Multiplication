
#include <hip/hip_runtime.h>
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk, int blk) {

    int bx = blockIdx.x;    int by = blockIdx.y;
    int tx = threadIdx.x;  int ty = threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*by;
    int bBegin = blockDim.y*bx;
    double Csub = 0;
    // Declaration of shared memory buffers 
    extern __shared__ double As[];
    double *Bs = (double *)As + (blk*blk);

    // Starting index of A and B for the thread
    int aInd = aBegin + Ni * tx + ty;
    int bInd = bBegin + Nj * ty + tx;

    int cBegin = bx*blockDim.x + by*blockDim.y*Nj;
    int cInd = cBegin + ty*Nj + tx;

    int aEnd = Ni*Nk;
    int bEnd = Nj*Nk;
    int cEnd = Ni*Nj;

    for (int kt = 0; kt < Nk; kt += blockDim.y) {

        if( aBegin + ty < Ni && aInd < aEnd ) {
            As[tx*blk + ty] = A[ aInd ];
            
        }
        else {
            As[tx*blk + ty] = 0;

        }

        if( bBegin + tx < Nj && bInd < bEnd ) {
            Bs[ty*blk + tx] = B[ bInd ];
        }
        else {
            Bs[ty*blk + tx] = 0;
        }

        __syncthreads();

        if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {
            for (int k = kt; k < kt + blockDim.y; ++k) { 
            
                Csub += As[(k - kt)*blk + ty] * Bs[(k - kt)*blk + tx];
            
            }
        }
        
        __syncthreads();
        aInd += Ni*blockDim.y;
        bInd += Nj*blockDim.y;
    }

    if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub;
    }
}