
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define threshold 0.00001

#define FIXME1 1
#define FIXME2 2
#define FIXME3 3
#define FIXME4 4


void checkCUDAError(const char *msg);

hipEvent_t start, stop;
float tstart, elapsedTime;

__global__ void atb(const int *A, const int *B, int *D, int*C, int Ni, int Nj, int Nk, int i, int j);

int main( int argc, char* argv[] ){

  int *h_A, *h_B, *h_C, *h_D, *h_Cref, *d_A, *d_B, *d_C, *d_D;
  int i,j,k,Ni,Nj,Nk;
  
  char* filename = argv[0];
  char* str2;
  str2 = ".txt";
  char * str3 = (char *) malloc(1 + strlen(filename)+ strlen(str2) );
  strcpy(str3, filename);
  strcat(str3, str2);

  FILE* fptr;
  fptr = fopen( str3, "w" );

  Ni = 4;
  Nj = 4;
  Nk = 8192;

  printf("Specify Matrix dimension Ni, Nj, Nk: %d %d %d", Ni, Nj, Nk);
  fprintf( fptr, "Specify Matrix dimension Ni, Nj, Nk: %d %d %d", Ni, Nj, Nk);
  
  // scanf("%d %d %d", &Ni,&Nj,&Nk);
  h_A = (int *) malloc(sizeof(int)*Ni*Nk);
  h_B = (int *) malloc(sizeof(int)*Nk*Nj);
  h_C = (int *) malloc(sizeof(int)*Ni*Nj);
  h_Cref = (int *) malloc(sizeof(int)*Ni*Nj);
  for (i=0; i<Ni; i++)
   for (k=0; k<Nk; k++)
    h_A[k*Ni+i] = 2;//i*Ni+i-1;
  for (k=0; k<Nk; k++)
   for (j=0; j<Nj; j++)
    h_B[k*Nj+j] = 2890;//j*Nj+j+1;
  for (i=0; i<Ni; i++)
   for (j=0; j<Nj; j++) {
    h_C[i*Nj+j] = 0;
    h_Cref[i*Nj+j] = 0;}

  for (i=0;i<Ni;i++)
   for (k=0;k<Nk;k++)
    for (j=0;j<Nj;j++)
  // h_Cref[i][j] += h_A[k][i]*h_B[k][j];
     h_Cref[i*Nj+j] += h_A[i+Ni*k]*h_B[k*Nj+j];
  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, Nk*Ni*sizeof(int));
  hipMalloc(&d_B, Nj*Nk*sizeof(int));
  hipMalloc(&d_C, Ni*Nj*sizeof(int));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, Nk*Ni*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Nj*Nk*sizeof(int), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D failure");

  int BSIZE = 64;
  int bsx = BSIZE;
  int bsy = 1;
  int gridsz = ceil( Nk/( (double)bsx ) );

  dim3 block(bsx, bsy);
  dim3 grid( gridsz, 1 );

    h_D = (int *) malloc(sizeof(int)*gridsz);
    for( int i = 0; i < gridsz; i++ ) {
        h_D[i] = 0;
    }
  hipMalloc(&d_D, gridsz*sizeof(int));
  hipMemcpy(d_D, h_D, gridsz*sizeof(int), hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  for(int trial=0;trial<5;trial++)
  {

   hipEventRecord(start);
   // Launch kernel

    for( int i = 0; i < Ni; i++ ) {
        for( int j = 0; j < Nj; j++ ) {

            

            atb<<<grid, block>>>(d_A, d_B, d_D, d_C, Ni,Nj,Nk, i, j);
            // cudaMemcpy(h_D, d_D, gridsz*sizeof(double), cudaMemcpyDeviceToHost);

            // double sum = 0;
            // for( int k = 0; k < gridsz; k++ ) {
            //     sum += h_D[k];
            // }

            // h_C[i*Nj + j] = sum;
        }
    }


   hipEventRecord(stop);
   checkCUDAError("kernel launch");
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start,stop);
//   cudaDeviceSynchronize();
   // Copy results back to host
   hipMemcpy(h_C, d_C, Ni*Nj*sizeof(int), hipMemcpyDeviceToHost);
   checkCUDAError("hipMemcpy D2H");
   for (int l = 0; l < Ni*Nj; l++) if (fabs((h_C[l] - h_Cref[l])/h_Cref[l])>threshold) {
    printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", l, h_C[l], h_Cref[l]); 
    fprintf(fptr, "Error: mismatch at linearized index %d, was: %f, should be: %f\n", l, h_C[l], h_Cref[l]);
    // return -1;
  }
   printf("<Ni=%d,Nj=%d,Nk=%d>: Trial %d: GFLOPS: %.2f\n",Ni,Nj,Nk,trial,2.0e-6*Ni*Nj*Nk/elapsedTime);
  }
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

