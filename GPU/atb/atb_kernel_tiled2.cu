
#include <hip/hip_runtime.h>
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    int bx = blockIdx.x;    int by = blockIdx.y;
    int tx = threadIdx.x;  int ty = threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*by;
    int bBegin = blockDim.y*bx;
    double Csub = 0;
    // Declaration of shared memory buffers 
    __shared__ double As[32][32];
    __shared__ double Bs[32][32];

    // Starting index of A and B for the thread
    int aInd = aBegin + Ni * tx + ty;
    int bInd = bBegin + Nj * ty + tx;

    int cBegin = bx*blockDim.x + by*blockDim.y*Nj;
    int cInd = cBegin + ty*Nj + tx;


    int aEnd = Ni*Nk;
    int bEnd = Nj*Nk;
    int cEnd = Ni*Nj;

    for (int kt = 0; kt < Nk; kt += blockDim.y) {

        if( aBegin + ty < Ni && aInd < aEnd ) {
            As[tx][ty] = A[ aInd ];
            
        }
        else {
            As[tx][ty] = 0;

        }

        if( bBegin + tx < Nj && bInd < bEnd ) {
            Bs[ty][tx] = B[ bInd ];
        }
        else {
            Bs[ty][tx] = 0;
        }

        __syncthreads();

        if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {
            for (int k = kt; k < kt + blockDim.y; ++k) { 
            
                Csub += As[k - kt][ty] * Bs[k - kt][tx];
            
            }
        }
        
        __syncthreads();
        aInd += Ni*blockDim.y;
        bInd += Nj*blockDim.y;
    }

    if( bx*blockDim.x + tx < Nj && cInd < cEnd ) {
        C[ cInd ] = Csub;
    }
}