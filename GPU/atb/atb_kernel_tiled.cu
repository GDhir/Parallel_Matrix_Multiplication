
#include <hip/hip_runtime.h>
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {

    int bx = blockIdx.x;    int by = blockIdx.y;
    int tx = threadIdx.x;  int ty = threadIdx.y;
    // Starting index of A and B for the thread block
    int aBegin = blockDim.y*by;
    int bBegin = blockDim.y*bx;
    double Csub = 0;
    // Declaration of shared memory buffers 
    __shared__ double As[2][2];
    __shared__ double Bs[2][2];

    // Starting index of A and B for the thread
    // int aInd = aBegin + Ni * tx + ty;
    // int bInd = bBegin + Nj * ty + tx;

    int acol = blockDim.y*by + ty;
    int arow = tx;
    int bcol = blockDim.y*bx + tx;
    int brow = ty;

    // int cInd = bx*blockDim.x + by*blockDim.y*Nj + ty*Nj + tx;

    int col = bx*blockDim.x + tx;
    int row = by*blockDim.y + ty;

    for (int kt = 0; kt < Nk; kt += blockDim.y) {

        if( acol < Ni && arow < Nk ) {
            As[tx][ty] = A[ arow*Ni + acol ];
            
        }
        else {
            As[tx][ty] = 0;

        }

        if( bcol < Nj && brow < Nk ) {
            Bs[ty][tx] = B[ brow*Nj + bcol ];
        }
        else {
            Bs[ty][tx] = 0;
        }

        __syncthreads();

        if( col < Nj && row < Ni ) {
            for (int k = kt; k < kt + blockDim.y; ++k) { 
            
                Csub += As[k - kt][ty] * Bs[k - kt][tx];
            
            }
        }
        
        __syncthreads();
        arow += blockDim.y;
        brow += blockDim.y;
    }
    if( col < Nj && row < Ni ) {
        C[ row*Nj + col ] = Csub;
    }
}