
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define threshold 0.0000001

#define FIXME1 1
#define FIXME2 2
#define FIXME3 3
#define FIXME4 4


void checkCUDAError(const char *msg);

hipEvent_t start, stop;
float tstart, elapsedTime;

__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atb_kernel_dot(const double *A, const double *B, double *D, double* C, int Ni, int Nj, int Nk, int i, int j);
__global__ void atb_kernel_tiled32(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atb_kernel_tiled_ijkunroll32(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atb_kernel_tiled_kunroll32(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atb_kernel_tiled32(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atb_kernel_slim(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atb_kernel_slimSM_kunroll8(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atb_kernel_slimSM_kunroll16(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atb_kernel_slimSM_kunroll32(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);
__global__ void atb_kernel_tiled_ijkunroll16(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);

void atb_seq(const double *__restrict__ A, const double *__restrict__ B, double *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[k][i]*B[k][j];
     C[i*Nj+j]=C[i*Nj+j]+A[k*Ni+i]*B[k*Nj+j];
}

void launcher( const double *d_A, const double *d_B, double *d_C, int Ni, int Nj, int Nk ) {

  int BSIZE = 32;
  int bsx;
  int bsy;

  if( Ni >= 180 && Nj >= 180 ) {

    if( Nk >= 32 ) {

      printf( "t1" );
      bsx = BSIZE;
      bsy = BSIZE;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)2*bsx ) ), ceil( Ni/( (double)2*bsy ) ) );
      atb_kernel_tiled_ijkunroll32<<<grid, block>>>(d_A, d_B, d_C, Ni, Nj, Nk);
    
    }
    else if( Nk >= 16 ) {
      printf( "t2" );
      bsx = BSIZE/2;
      bsy = BSIZE/2;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)2*bsx ) ), ceil( Ni/( (double)2*bsy ) ) );
      atb_kernel_tiled_ijkunroll16<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);

    }
    else if( Nk >= 8 ) {
      printf( "t3" );
      bsx = BSIZE/4;
      bsy = 1;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
      atb_kernel_slimSM_kunroll8<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);

    }
    else {

      bsx = 4;
      bsy = 1;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
      atb_kernel_slim<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);

    }

  }
  else if( Ni > 128 && Nk > 128 ) {

    if( Nj >= 512 ) {
      printf( "t4" );
      bsx = BSIZE;
      bsy = BSIZE;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)2*bsx ) ), ceil( Ni/( (double)2*bsy ) ) );
      atb_kernel_tiled_ijkunroll32<<<grid, block>>>(d_A, d_B, d_C, Ni, Nj, Nk);

    }
    else {
      printf( "t5" );
      bsx = BSIZE;
      bsy = 1;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
      atb_kernel_slimSM_kunroll32<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);

    }

  }
  else if( Ni > 64 && Nj > 256 && Nk < 1024 ) {
    printf( "t6" );
    bsx = BSIZE;
    bsy = BSIZE;

    dim3 block(bsx, bsy);
    dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
    atb_kernel_tiled_kunroll32<<<grid, block>>>(d_A, d_B, d_C, Ni, Nj, Nk);

  }
  else if( Ni > 64 && Nj > 1024 ) {

    if( Nk < 2048 ) {
      printf( "t7" );

      if( Nk > 16 ) {

        bsx = BSIZE;
        bsy = BSIZE;

        dim3 block(bsx, bsy);
        dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
        atb_kernel_tiled_kunroll32<<<grid, block>>>(d_A, d_B, d_C, Ni, Nj, Nk);
      }
      else {

        bsx = BSIZE/4;
        bsy = 1;

        dim3 block(bsx, bsy);
        dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
        atb_kernel_slimSM_kunroll8<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);

      }
    }
    else {
      printf( "t8" );
      bsx = BSIZE;
      bsy = BSIZE;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)2*bsx ) ), ceil( Ni/( (double)2*bsy ) ) );
      atb_kernel_tiled_ijkunroll32<<<grid, block>>>(d_A, d_B, d_C, Ni, Nj, Nk);

    }

  }
  else if( Ni > 32 && Nj > 1024 && Nk > 1024 ) {
    printf( "t10" );
      bsx = BSIZE;
      bsy = BSIZE;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)2*bsx ) ), ceil( Ni/( (double)2*bsy ) ) );
      atb_kernel_tiled_ijkunroll32<<<grid, block>>>(d_A, d_B, d_C, Ni, Nj, Nk);

  }
  else if( Ni > 32 && Nj > 4096 && Nk > 256 ) {
    printf( "t11" );
      bsx = BSIZE;
      bsy = BSIZE;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
      atb_kernel_tiled_kunroll32<<<grid, block>>>(d_A, d_B, d_C, Ni, Nj, Nk);

  }
  else {

    if( Ni >= 64 && Nj >= 256 && Nk >= 32 ) {
      bsx = BSIZE;
      bsy = 1;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
      atb_kernel_slimSM_kunroll32<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);
    }
    else if( Ni < 4 || Nj < 4  ) {
      bsx = 2;
      bsy = 1;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
      atb_kernel_slim<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);
    }
    else if( Ni < 8 || Nj < 8 ) {

      bsx = 4;
      bsy = 1;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
      atb_kernel_slim<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);

    }
    else if( Ni < 16 || Nj < 16  ) {

      bsx = 8;
      bsy = 1;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
      atb_kernel_slimSM_kunroll8<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);

    }
    else if( Ni < 32 || Nj < 32  ) {

      bsx = 16;
      bsy = 1;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
      atb_kernel_slimSM_kunroll16<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);

    }
    else {

      bsx = 32;
      bsy = 1;

      dim3 block(bsx, bsy);
      dim3 grid( ceil( Nj/( (double)bsx ) ), ceil( Ni/( (double)bsy ) ) );
      atb_kernel_slimSM_kunroll32<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);

    }

}


};

int main(){

  double *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k,Ni,Nj,Nk;

  printf("Specify Matrix dimension Ni, Nj, Nk: ");
  scanf("%d %d %d", &Ni,&Nj,&Nk);
  h_A = (double *) malloc(sizeof(double)*Ni*Nk);
  h_B = (double *) malloc(sizeof(double)*Nk*Nj);
  h_C = (double *) malloc(sizeof(double)*Ni*Nj);
  h_Cref = (double *) malloc(sizeof(double)*Ni*Nj);
  for (i=0; i<Ni; i++)
   for (k=0; k<Nk; k++)
    h_A[k*Ni+i] = k*Ni+i-1;
  for (k=0; k<Nk; k++)
   for (j=0; j<Nj; j++)
    h_B[k*Nj+j] = k*Nj+j+1;
  for (i=0; i<Ni; i++)
   for (j=0; j<Nj; j++) {
    h_C[i*Nj+j] = 0;
    h_Cref[i*Nj+j] = 0;}

  for (i=0;i<Ni;i++)
   for (k=0;k<Nk;k++)
    for (j=0;j<Nj;j++)
  // h_Cref[i][j] += h_A[k][i]*h_B[k][j];
     h_Cref[i*Nj+j] += h_A[i+Ni*k]*h_B[k*Nj+j];
  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, Nk*Ni*sizeof(double));
  hipMalloc(&d_B, Nj*Nk*sizeof(double));
  hipMalloc(&d_C, Ni*Nj*sizeof(double));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, Nk*Ni*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Nj*Nk*sizeof(double), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D failure");

  // dim3 block(FIXME1,FIXME2);
  // dim3 grid(FIXME3,FIXME4);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  for(int trial=0;trial<5;trial++)
  {
   hipEventRecord(start);
   // Launch kernel
   launcher(d_A, d_B, d_C, Ni, Nj, Nk);
  //  atb<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);
   hipEventRecord(stop);
   checkCUDAError("kernel launch");
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start,stop);
//   cudaDeviceSynchronize();
   // Copy results back to host
   hipMemcpy(h_C, d_C, Ni*Nj*sizeof(double), hipMemcpyDeviceToHost);
   checkCUDAError("hipMemcpy D2H");
   for (int l = 0; l < Ni*Nj; l++) if (fabs((h_C[l] - h_Cref[l])/h_Cref[l])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", l, h_C[l], h_Cref[l]); return -1;}
   printf("<Ni=%d,Nj=%d,Nk=%d>: Trial %d: GFLOPS: %.2f\n",Ni,Nj,Nk,trial,2.0e-6*Ni*Nj*Nk/elapsedTime);
  }
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


