
#include <hip/hip_runtime.h>
#define BSIZE 16

// matrix multiply kernel: C = A^T * B
__global__ void atb_kernel_slimSM_kunroll16(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness

        int tx = threadIdx.x;
        int i = blockIdx.y;
        int j = blockIdx.x*blockDim.x+tx;
        int idx = tx*Ni + i;
        int remk;

        int aEnd = Nk*Ni;
        
        __shared__ double As[BSIZE];

        double sum = 0;

        for( int ks = 0; ks < Nk; ks += BSIZE ) {

            if( idx + ks*Ni < aEnd )
                As[tx] = A[ ks*Ni + idx ];
            else
                As[tx] = 0;

            __syncthreads();


            if( j < Nj ) {
                for(int k = ks; k < ks + remk; k++ ) {
                    sum += As[k - ks]*B[ k*Nj + j ];
                }

                for (int k = ks + remk; k < ks + remk + BSIZE; k += 2) {
                    sum += As[k - ks]*B[ k*Nj + j ];
                    sum += As[k + 1 - ks]*B[ (k + 1)*Nj + j ]; 
                }
            }

            __syncthreads();

        }
            
        if( i < Ni && j < Nj ) {
            C[ i * Nj + j ] = sum;
        }
    
    }
    
    