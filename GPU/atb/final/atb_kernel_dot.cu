
#include <hip/hip_runtime.h>
#define BSIZE 512

__global__ void atb_kernel_dot(const double *A, const double *B, double *D, double* C, int Ni, int Nj, int Nk, int i, int j) {

    int bx = blockIdx.x;
    int tx = threadIdx.x;

    __shared__ double cache[ BSIZE ];

    int kId = bx*blockDim.x + tx;

    if( kId < Nk ) {
        cache[ tx ] = A[ kId*Ni + i ]*B[ kId*Nj + j ];
    }
    else {
        cache[ tx ] = 0;
    }

    int sz = ceil( BSIZE/2.0 );

    __syncthreads();

    while( sz > 1 ) {

        if( tx < sz ) {

            cache[tx] += cache[tx + sz];
            cache[tx + sz] = 0;

        }

        sz = ceil( sz/2.0 );

        __syncthreads();

    }

    if( tx == 0 ) {

        D[bx] = cache[0] + cache[1];
        // atomicAdd1( &C[ i*Nj + j ], D[bx] );
    }

}