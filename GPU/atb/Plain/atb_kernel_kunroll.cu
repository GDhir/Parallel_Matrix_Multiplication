
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int i = blockIdx.y*blockDim.y+threadIdx.y;
        int j = blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i < Ni ) && ( j < Nj ) ) { 
            double sum = 0;
    
            int remk = Nk%2;
            
            for( int k = 0; k < remk; k++ ) {
                sum += A[ k*Ni + i ]*B[ k*Nj + j ]; 
            }

            for (int k = remk; k < Nk; k += 2) {
                 sum += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                 sum += A[ (k + 1)*Ni + i ]*B[ (k + 1)*Nj + j ]; 
            }
                 
            C[ i * Nj + j ] = sum;
        }
    
    }
    
    