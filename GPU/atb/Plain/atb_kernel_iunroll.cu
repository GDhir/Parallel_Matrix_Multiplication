
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int i = 2*( blockIdx.y*blockDim.y+threadIdx.y );
        int j = blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i < Ni ) && ( j < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
    
            if( i + 1 < Ni ) {
                for (int k = 0; k < Nk; ++k) {
                    sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                    sum1 += A[ k*Ni + i + 1 ]*B[ k*Nj + j ]; 
                }
            }
            else {
                for (int k = 0; k < Nk; ++k) {
                    sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                }
            }
                 
            C[ i * Nj + j ] = sum0;

            if( i + 1 < Ni )
                C[ ( i + 1 ) * Nj + j ] = sum1;
        }
    
    }
    
    