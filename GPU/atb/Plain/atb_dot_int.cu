
#include <hip/hip_runtime.h>
#define BSIZE 64

__global__ void atb(const int *A, const int *B, int *D, int* C, int Ni, int Nj, int Nk, int i, int j) {

    int bx = blockIdx.x;
    int tx = threadIdx.x;

    __shared__ int cache[ BSIZE ];

    int kId = bx*blockDim.x + tx;

    if( kId < Nk ) {
        cache[ tx ] = A[ kId*Ni + i ]*B[ kId*Nj + j ];
    }
    else {
        cache[ tx ] = 0;
    }

    int sz = ceil( BSIZE/2.0 );

    __syncthreads();

    while( sz > 1 ) {

        if( tx < sz ) {

            cache[tx] += cache[tx + sz];
            cache[tx + sz] = 0;

        }

        sz = ceil( sz/2.0 );

        __syncthreads();

    }

    if( tx == 0 ) {

        D[bx] = cache[0] + cache[1];
        // atomicAdd1( &C[ i*Nj + j ], D[bx] );
    }

    __syncthreads();

    if( bx == 0 ) {

        while( kId + blockDim.x < gridDim.x ) {

            D[tx] += D[kId + blockDim.x];
            kId += blockDim.x;

        }


        if( gridDim.x >= blockDim.x )
            sz = ceil( blockDim.x/2.0 );
        else
            sz = ceil( gridDim.x/2.0 );

        __syncthreads();

        while( sz > 1 ) {

            if( tx < sz ) {

                D[tx] += D[tx + sz];
                D[tx + sz] = 0;

            }

            sz = ceil( sz/2.0 );

            __syncthreads();

        }


        if( tx == 0 ) {

            C[i*Nj + j] = D[0] + D[1];
            
        }

    }
}