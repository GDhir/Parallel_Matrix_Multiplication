
#include <hip/hip_runtime.h>
#define BSIZE 64

__device__ double atomicAdd1(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void atb(const double *A, const double *B, double *D, double* C, int Ni, int Nj, int Nk, int i, int j) {

    int bx = 2*blockIdx.x;
    int tx = threadIdx.x;

    __shared__ double cache[ 2*BSIZE ];

    int kId = bx*blockDim.x + tx;

    if( kId + BSIZE < Nk ) {
        cache[ tx ] = A[ kId*Ni + i ]*B[ kId*Nj + j ];
        cache[ tx + BSIZE ] = A[ (kId + BSIZE)*Ni + i ]*B[ (kId + BSIZE)*Nj + j ];
    }
    else if( kId < Nk ) {
        cache[ tx ] = A[ kId*Ni + i ]*B[ kId*Nj + j ];
        cache[ tx + BSIZE ] = 0;
    }
    else {
        cache[ tx ] = 0;
        cache[ tx + BSIZE ] = 0;
    }

    int sz = BSIZE;

    __syncthreads();

    while( sz > 1 ) {

        if( tx < sz ) {

            cache[tx] += cache[tx + sz];
            cache[tx + sz] = 0;

        }

        sz = ceil( sz/2.0 );

        __syncthreads();

    }

    if( tx == 0 ) {

        D[bx] = cache[0] + cache[1];
        // atomicAdd1( &C[ i*Nj + j ], D[bx] );
    }

    __syncthreads();

    if( bx == 0 ) {

        while( kId + blockDim.x < gridDim.x ) {

            D[tx] += D[kId + blockDim.x];
            kId += blockDim.x;

        }


        if( gridDim.x >= blockDim.x )
            sz = ceil( blockDim.x/2.0 );
        else
            sz = ceil( gridDim.x/2.0 );

        __syncthreads();

        while( sz > 1 ) {

            if( tx < sz ) {

                D[tx] += D[tx + sz];
                D[tx + sz] = 0;

            }

            sz = ceil( sz/2.0 );

            __syncthreads();

        }


        if( tx == 0 ) {

            C[i*Nj + j] = D[0] + D[1];
            
        }

        __syncthreads();

    }
}