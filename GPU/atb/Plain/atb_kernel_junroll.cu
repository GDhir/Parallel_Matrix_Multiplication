
#include <hip/hip_runtime.h>
// matrix multiply kernel: C = A^T * B
__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk) {
    // Initially empty; will clearly not pass correctness
    
        int i = blockIdx.y*blockDim.y+threadIdx.y;
        int j = 2*blockIdx.x*blockDim.x+threadIdx.x;
        
        if ( ( i < Ni ) && ( j < Nj ) ) { 
            double sum0 = 0;
            double sum1 = 0;
    
            if( j + blockDim.x < Nj ) {
                for (int k = 0; k < Nk; ++k) {
                    sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                    sum1 += A[ k*Ni + i ]*B[ k*Nj + j + blockDim.x ]; 
                }
            }
            else {

                for (int k = 0; k < Nk; ++k) {
                    sum0 += A[ k*Ni + i ]*B[ k*Nj + j ]; 
                }

            }

            C[ i * Nj + j ] = sum0;

            if( j + blockDim.x < Nj ) {
                C[ i * Nj + j + blockDim.x ] = sum1;
            }
        }
    
    }
    
    